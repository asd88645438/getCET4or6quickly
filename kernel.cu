#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "../common/book.h"
#include "../common/cpu_anim.h"

#define DIM 1024
#define PI 3.1415926535897932f

__global__ void kernel(int* dev_init, int* dev_output) {

		int alive = 0;
		if (blockIdx.y - 1 >= 0)
		{
			if (dev_init[(blockIdx.y - 1)*(DIM / 16) + blockIdx.x] == 1) alive++;

			if (blockIdx.x + 1 < (DIM / 16))
			{
				if (dev_init[(blockIdx.y - 1)*(DIM / 16) + blockIdx.x + 1] == 1) alive++;
			}
		}
		if (blockIdx.y + 1 < (DIM / 16))
		{
			if (dev_init[(blockIdx.y + 1)*(DIM / 16) + blockIdx.x] == 1) alive++;

			if (blockIdx.x - 1 >= 0)
			{
				if (dev_init[(blockIdx.y + 1)*(DIM / 16) + blockIdx.x - 1] == 1) alive++;
			}
		}
		if (blockIdx.x - 1 >= 0)
		{
			if (dev_init[blockIdx.y*(DIM / 16) + blockIdx.x - 1] == 1) alive++;

			if (blockIdx.y - 1 >= 0)
			{
				if (dev_init[(blockIdx.y - 1)*(DIM / 16) + blockIdx.x - 1] == 1) alive++;
			}
		}
		if (blockIdx.x + 1 < (DIM / 16))
		{
			if (dev_init[blockIdx.y*(DIM / 16) + blockIdx.x + 1] == 1) alive++;

			if (blockIdx.y + 1 < (DIM / 16))
			{
				if (dev_init[(blockIdx.y + 1)*(DIM / 16) + blockIdx.x + 1] == 1) alive++;
			}
		}

		//�ж�alive����
		if (dev_output[blockIdx.y*(DIM / 16) + blockIdx.x] == 1) {
			if (alive < 2)
			{
				dev_output[blockIdx.y*(DIM / 16) + blockIdx.x] = 0;
			}
			else if (alive < 4)
			{
				dev_output[blockIdx.y*(DIM / 16) + blockIdx.x] = 1;
			}
			else
			{
				dev_output[blockIdx.y*(DIM / 16) + blockIdx.x] = 0;
			}
		}
		else if (dev_output[blockIdx.y*(DIM / 16) + blockIdx.x] == 0)
		{
			if (alive == 3)
			{
				dev_output[blockIdx.y*(DIM / 16) + blockIdx.x] = 1;
			}
		}
	

}

__global__ void kernelShow(unsigned char *ptr, int ticks, int* dev_init)
{
	//�������ص�����
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	__shared__ int  shared[16][16];

	//���Ƶ�ǰͼ��
	if (dev_init[blockIdx.y*(DIM / 16) + blockIdx.x] == 1)
		shared[threadIdx.x][threadIdx.y] = 1;
	else
		shared[threadIdx.x][threadIdx.y] = 0;
	__syncthreads();

	//����߿�
	if (threadIdx.x == 0 || threadIdx.y == 0 || threadIdx.x == 15 || threadIdx.y == 15)
		shared[threadIdx.x][threadIdx.y] = 0;

	//���Ʒ��鲨�ƶ�̬
	float fx = x - DIM / 2;
	float fy = y - DIM / 2;
	float d = sqrtf(fx * fx + fy * fy);
	unsigned char grey = (unsigned char)(128.0f + 127.0f *
		cos(d / 10.0f - ticks / 7.0f) /
		(d / 10.0f + 1.0f));
	
	//���
	ptr[offset * 4 + 0] = shared[15 - threadIdx.x][15 - threadIdx.y] * 255;
	ptr[offset * 4 + 1] = shared[15 - threadIdx.x][15 - threadIdx.y] * grey;
	ptr[offset * 4 + 2] = 0;
	ptr[offset * 4 + 3] = 255;


}


struct DataBlock {
	unsigned char   *dev_bitmap;
	CPUAnimBitmap  *bitmap;
	int* dev_init;
	int* dev_output;
};

void generate_frame(DataBlock *d, int ticks) {

	dim3    blocks(DIM / 16, DIM / 16);
	dim3    threads(16, 16);

	//����ͼ��
	kernelShow << <blocks, threads >> >(d->dev_bitmap, ticks, d->dev_init);

	HANDLE_ERROR(hipMemcpy(d->bitmap->get_ptr(),
		d->dev_bitmap,
		d->bitmap->image_size(),
		hipMemcpyDeviceToHost));

	if (ticks > 100) {
		if (ticks % 10 == 0)
		{
			kernel << <blocks, 1 >> > (d->dev_init, d->dev_output);
			HANDLE_ERROR(hipMemcpy(d->dev_init, d->dev_output, DIM / 16 * DIM / 16 * sizeof(int), hipMemcpyDeviceToDevice));
		}
	}

}

// clean up memory allocated on the GPU
void cleanup(DataBlock *d) {
	HANDLE_ERROR(hipFree(d->dev_bitmap));
	HANDLE_ERROR(hipFree(d->dev_init));
	HANDLE_ERROR(hipFree(d->dev_output));
}

int main(void) {
	DataBlock   data;
	CPUAnimBitmap  bitmap(DIM, DIM, &data);
	data.bitmap = &bitmap;

	int  *initdata;
	initdata = new int[(DIM / 16) *(DIM / 16)];
	memset(initdata, 0, (DIM / 16) * (DIM / 16) * sizeof(int));
	HANDLE_ERROR(hipMalloc((void**)&data.dev_bitmap, bitmap.image_size()));


	//��ʼ������
	/*initdata[(16)*(DIM / 16) + (16)] = 1;
	initdata[(16)*(DIM / 16) + (15)] = 1;
	initdata[(16)*(DIM / 16) + (17)] = 1;
	initdata[(15)*(DIM / 16) + (16)] = 1;
	initdata[(17)*(DIM / 16) + (16)] = 1;
	initdata[(17)*(DIM / 16) + (17)] = 1;
	initdata[(15)*(DIM / 16) + (15)] = 1;
	initdata[(15)*(DIM / 16) + (17)] = 1;
	initdata[(17)*(DIM / 16) + (15)] = 1;*/   //9X9������

	/*initdata[(16)*(DIM / 16) + (16)] = 1;
	initdata[(16)*(DIM / 16) + (15)] = 1;
	initdata[(16)*(DIM / 16) + (17)] = 1;
	initdata[(15)*(DIM / 16) + (16)] = 1;
	initdata[(17)*(DIM / 16) + (16)] = 1;*/     //ʮ��


	initdata[(32)*(DIM / 16) + (16)] = 1;
	initdata[(32)*(DIM / 16) + (17)] = 1;
	initdata[(32)*(DIM / 16) + (18)] = 1;
	initdata[(32)*(DIM / 16) + (19)] = 1;
	initdata[(32)*(DIM / 16) + (20)] = 1;
	initdata[(32)*(DIM / 16) + (21)] = 1;
	initdata[(32)*(DIM / 16) + (22)] = 1;
	initdata[(32)*(DIM / 16) + (23)] = 1;
	initdata[(32)*(DIM / 16) + (24)] = 1;

	initdata[(32)*(DIM / 16) + (29)] = 1;
	initdata[(32)*(DIM / 16) + (30)] = 1;
	initdata[(32)*(DIM / 16) + (31)] = 1;
	initdata[(32)*(DIM / 16) + (32)] = 1;
	initdata[(32)*(DIM / 16) + (33)] = 1;
	initdata[(32)*(DIM / 16) + (34)] = 1;
	initdata[(32)*(DIM / 16) + (35)] = 1;
	initdata[(32)*(DIM / 16) + (36)] = 1;
	initdata[(32)*(DIM / 16) + (37)] = 1;
	initdata[(32)*(DIM / 16) + (38)] = 1;

	initdata[(32)*(DIM / 16) + (44)] = 1;
	initdata[(32)*(DIM / 16) + (45)] = 1;
	initdata[(32)*(DIM / 16) + (46)] = 1;


	HANDLE_ERROR(hipMalloc((void**)&data.dev_init, DIM / 16 * DIM / 16 * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&data.dev_output, DIM / 16 * DIM / 16 * sizeof(int)));

	HANDLE_ERROR(hipMemcpy(data.dev_init, initdata, DIM / 16 * DIM / 16 * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(data.dev_output, initdata, DIM / 16 * DIM / 16 * sizeof(int), hipMemcpyHostToDevice));

	bitmap.anim_and_exit((void(*)(void*, int))generate_frame,
		(void(*)(void*))cleanup);
}
